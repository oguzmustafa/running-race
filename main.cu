#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#include ""
#include "./runner.cuh"

#include<windows.h>

using namespace std;

__global__ void init(unsigned int seed, hiprandState_t* states) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, threadIdx.x, 0, &states[i]);
}

__global__ void randoms(hiprandState_t* states, int* speed) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	speed[i] = hiprand(&states[i]) % 5;
	speed[i] = speed[i] + 1;
}

__global__ void race(int *speed, int *location,int *de) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (location[i] < 100) {
		location[i] = speed[i] + location[i];
	}
	if (location[i] >= 100) {
		//location[i] = 0;
		speed[i] = 0;
		de[0] += 1;
		location[i] = de[0];
	}
}

int main() {
	hiprandState_t* states;
	hipMalloc((void**)&states, NN * sizeof(hiprandState_t));
	init << <1, NN >> > (time(0), states);

	int speed[NN];
	int* de_speed;

	hipMalloc((void**)&de_speed, NN * sizeof(int));
	randoms << <1,NN >> > (states, de_speed);
	hipMemcpy(speed, de_speed, NN * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < NN; i++) {
		r[i].speed = speed[i];
	}

	int location[NN];
	int *de_location;
	hipMalloc((void**)&de_location, NN * sizeof(int));

	int *de;
	int d[1];

	hipMalloc((void**)&de, 1 * sizeof(int));

	int sec = 0;
	for (sec; sec < NN; sec++) {
		race << <1, NN >> > (de_speed, de_location, de);
		hipMemcpy(d, de, 1 * sizeof(int), hipMemcpyDeviceToHost);
		if (d[0] > 0) {
			break;
		}
		Sleep(1000);
	}

	hipMemcpy(location, de_location, NN * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < NN; i++) {
		if (location[i] >= 0 && location[i] < 20){
			location[i] = p.length;
		}
		r[i].indis= i+1;
		r[i].location = location[i];
		r[i].p_location();
	}

	for (sec; sec < NN; sec++) {
		race << <1, NN >> > (de_speed, de_location, de);
		Sleep(1000);
	}

	hipMemcpy(location, de_location, NN * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < NN; i++) {
		r[i].rank = location[i];
		r[i].p_rank();
	}

	hipFree(de_location);
	hipFree(de_speed);
	hipFree(de);
	hipFree(states);

	return 0;
}